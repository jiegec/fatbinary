// taken from https://llvm.org/docs/CompileCudaWithLLVM.html
// https://gist.github.com/anonymous/855e277884eb6b388cd2f00d956c2fd4


#include <hip/hip_runtime.h>
#include <iostream>

__global__ void axpy(float a, float* x, float* y) {
  y[threadIdx.x] = a * x[threadIdx.x];
}

int main(int argc, char* argv[]) {
  const int kDataLen = 4;

  float a = 2.0f;
  float host_x[kDataLen] = {1.0f, 2.0f, 3.0f, 4.0f};
  float host_y[kDataLen];

  // Copy input data to device.
  float* device_x;
  float* device_y;
  hipMalloc(&device_x, kDataLen * sizeof(float));
  hipMalloc(&device_y, kDataLen * sizeof(float));
  hipMemcpy(device_x, host_x, kDataLen * sizeof(float),
             hipMemcpyHostToDevice);

  // Launch the kernel.
  axpy<<<1, kDataLen>>>(a, device_x, device_y);

  // Copy output data to host.
  hipDeviceSynchronize();
  hipMemcpy(host_y, device_y, kDataLen * sizeof(float),
             hipMemcpyDeviceToHost);

  // Print the results.
  for (int i = 0; i < kDataLen; ++i) {
    std::cout << "y[" << i << "] = " << host_y[i] << "\n";
  }

  hipDeviceReset();
  return 0;
}